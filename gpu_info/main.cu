// main.cu
#include <iostream>
#include "gpu_info.cuh"

// Function to identify the GPU
void identifyGPU() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cout << "No CUDA-capable devices found." << std::endl;
    } else {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        std::cout << "CUDA-capable device found!" << std::endl;
        std::cout << "Device Name: " << deviceProp.name << std::endl;
        std::cout << "Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "Total Global Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
    }
}

int main() {
    identifyGPU();
    return 0;
}
